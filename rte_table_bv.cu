#include "hip/hip_runtime.h"
#include "rte_table_bv.h"
#include <hip/hip_runtime.h>

#ifdef __cplusplus
extern "C" {
#endif

#include "rte_bv.h"
#include <rte_log.h>
#include <rte_malloc.h>
#include <stdlib.h>

#ifdef RTE_TABLE_STATS_COLLECT

#define RTE_TABLE_BV_STATS_PKTS_IN_ADD(table, val) table->stats.n_pkts_in += val
#define RTE_TABLE_BV_STATS_PKTS_LOOKUP_MISS(table, val) table->stats.n_pkts_lookup_miss += val

#else

#define RTE_TABLE_BV_STATS_PKTS_IN_ADD(table, val)
#define RTE_TABLE_BV_STATS_PKTS_LOOKUP_MISS(table, val)

#endif

struct rte_table_bv {
    uint32_t num_fields;
    struct rte_table_stats stats;
    const struct rte_table_bv_field_def *field_defs;

    uint8_t act_buf; // which double buffer is currently active
    uint32_t **ranges; // size==[num_fields][2*RTE_TABLE_BV_MAX_RANGES]
    uint32_t **bvs; // size==[num_fields][RTE_TABLE_BV_BS*2*RTE_TABLE_BV_MAX_RANGES]

    size_t *num_ranges;
    uint32_t *field_ptype_masks;
    uint32_t *field_offsets;
    uint8_t *field_sizes;

    uint32_t **ranges_dev;
    uint32_t **bvs_dev;

    uint8_t **pkts_data;
    uint8_t **pkts_data_h;

    uint32_t *packet_types;
    uint32_t *packet_types_h;

    rte_bv_markers_t *bv_markers; // size==num_fields
};

static inline int is_error(hipError_t e, const char *file, int line) {
    if(e!=hipSuccess) {
        fprintf(stderr, "[rte_table_bv] error: %s in %s (line %d)\n", hipGetErrorString(e), file, line);
        return 1;
    }
    return 0;
}

static int rte_table_bv_free(void *t_r) {
    if(t_r==NULL)
        return 0;

    struct rte_table_bv *t=(struct rte_table_bv *) t_r;

    for(size_t i=0; i<t->num_fields; ++i) {
        hipFree(t->ranges[i]);
        hipFree(t->bvs[i]);
    }
    hipFree(t->ranges_dev);
    hipFree(t->bvs_dev);

    hipFree(t->num_ranges);
    hipFree(t->field_offsets);
    hipFree(t->field_ptype_masks);
    hipFree(t->field_sizes);
	
	hipHostFree(t->pkts_data_h);
	hipHostFree(t->packet_types_h);

    for(uint32_t i=0; i<t->num_fields; ++i)
        rte_bv_markers_free(t->bv_markers+i);

    rte_free(t->bv_markers);
    rte_free(t->ranges);
    rte_free(t->bvs);

    rte_free(t);

    return 0;
}

#define IS_ERROR(X) is_error(X, __FILE__, __LINE__)

static void *rte_table_bv_create(void *params, int socket_id, uint32_t entry_size) {
    struct rte_table_bv_params *p=(struct rte_table_bv_params *) params;
    struct rte_table_bv *t=(struct rte_table_bv *) rte_malloc("t", sizeof(struct rte_table_bv), 0);
    memset(t, 0, sizeof(struct rte_table_bv));

    t->num_fields=p->num_fields;
    t->field_defs=p->field_defs;
    t->act_buf=0;

    t->ranges=(uint32_t **) rte_malloc("ranges_db", sizeof(uint32_t *)*t->num_fields, 0);
    t->bvs=(uint32_t **) rte_malloc("bvs_db", sizeof(uint32_t *)*t->num_fields, 0);

#define CHECK(X) if(IS_ERROR(X)) return NULL

    CHECK(hipHostAlloc((void **) &t->pkts_data_h, sizeof(uint8_t*)*RTE_TABLE_BV_MAX_PKTS, hipHostMallocMapped|hipHostMallocWriteCombined));
    CHECK(hipHostGetDevicePointer((void **) &t->pkts_data, t->pkts_data_h, 0));

    CHECK(hipHostAlloc((void **) &t->packet_types_h, sizeof(uint32_t)*RTE_TABLE_BV_MAX_PKTS, hipHostMallocMapped|hipHostMallocWriteCombined));
    CHECK(hipHostGetDevicePointer((void **) &t->packet_types, t->packet_types_h, 0));

    CHECK(hipMalloc((void **) &t->ranges_dev, sizeof(uint32_t *)*t->num_fields));
    CHECK(hipMalloc((void **) &t->bvs_dev, sizeof(uint32_t *)*t->num_fields));
    CHECK(hipMalloc((void **) &t->field_offsets, sizeof(uint32_t)*t->num_fields));
    CHECK(hipMalloc((void **) &t->field_ptype_masks, sizeof(uint32_t)*t->num_fields));
    CHECK(hipMalloc((void **) &t->field_sizes, sizeof(uint32_t)*t->num_fields));
    CHECK(hipMalloc((void **) &t->num_ranges, sizeof(uint64_t)*t->num_fields));

    for(size_t i=0; i<t->num_fields; ++i) {
        CHECK(hipMemcpy(t->field_offsets+i, &t->field_defs[i].offset, sizeof(uint32_t), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(t->field_sizes+i, &t->field_defs[i].size, sizeof(uint32_t), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(t->field_ptype_masks+i, &t->field_defs[i].ptype_mask, sizeof(uint32_t), hipMemcpyHostToDevice));

        CHECK(hipMalloc((void **) &t->ranges[i], sizeof(uint32_t)*((size_t) RTE_TABLE_BV_MAX_RANGES) *2));
        CHECK(hipMalloc((void **) &t->bvs[i], sizeof(uint32_t)*((size_t) RTE_TABLE_BV_BS) * ((size_t ) RTE_TABLE_BV_MAX_RANGES) *2));
    }

    CHECK(hipMemcpy(t->ranges_dev, t->ranges, sizeof(uint32_t *)*t->num_fields, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(t->bvs_dev, t->bvs, sizeof(uint32_t *)*t->num_fields, hipMemcpyHostToDevice));
#undef CHECK

    t->bv_markers=(rte_bv_markers_t *) rte_malloc("bv_markers", sizeof(rte_bv_markers_t)*t->num_fields, 0);

    for(size_t i=0; i<t->num_fields; ++i) {
        if(rte_bv_markers_create(&t->bv_markers[i])) {
            rte_table_bv_free(t);
            rte_log(RTE_LOG_ERR, RTE_LOGTYPE_HASH, "Error creating marker!\n");
            return NULL;
        }
    }

    return (void *) t;
}
#undef IS_ERROR

static inline void cal_from_to(uint32_t *from_to, uint32_t *v, uint8_t type, uint8_t size) {
    if(type==RTE_TABLE_BV_FIELD_TYPE_RANGE) {
        from_to[0]=*v;
        from_to[1]=v[1];
    } else {
        from_to[0]=(*v)&v[1];
        switch(size) {
        case 1:
            from_to[1]=(*v)|((uint8_t) (~v[1]));
            break;
        case 2:
            from_to[1]=(*v)|((uint16_t) (~v[1]));
            break;
        case 4:
            from_to[1]=(*v)|((uint32_t) (~v[1]));
            break;
        default:
#ifdef DEBUG
            fprintf(stderr, "[cal_from_to] error: unkown size: %d bits\n", size);
#endif
            break;
        }
    }
}

static int rte_table_bv_entry_add(void *t_r, void *k_r, void *e_r, int *key_found, void **e_ptr) {
    struct rte_table_bv *t=(struct rte_table_bv *) t_r;
    struct rte_table_bv_key *k=(struct rte_table_bv_key *) k_r;
    uint32_t *pos=(uint32_t *) e_r;
    *key_found=0;

    uint32_t from_to[2];
    rte_bv_ranges_t ranges;

    for(uint32_t f=0; f<t->num_fields; ++f) {
        cal_from_to(from_to, k->buf +(f<<1), t->field_defs[f].type, t->field_defs[f].size);
        rte_bv_markers_range_add(t->bv_markers+f, from_to, *pos);

        memset(&ranges, 0, sizeof(rte_bv_ranges_t));
        ranges.bv_bs=RTE_TABLE_BV_BS;
        ranges.ranges=t->ranges[t->num_fields+f];
        ranges.bvs=t->bvs[t->num_fields+f];
        rte_bv_markers_to_ranges(t->bv_markers+f, 1, sizeof(uint32_t), &ranges);
        hipMemcpy(t->num_ranges+f, (void *) &ranges.num_ranges, sizeof(uint64_t), hipMemcpyHostToDevice);
    }

    return 0;
}

static int rte_table_bv_entry_delete(void  *t_r, void *k_r, int *key_found, void *e) {
    struct rte_table_bv *t=(struct rte_table_bv *) t_r;
    struct rte_table_bv_key *k=(struct rte_table_bv_key *) k_r;
    *key_found=0;

    uint32_t from_to[2];
    rte_bv_ranges_t ranges;

    for(uint32_t f=0; f<t->num_fields; ++f) {
        cal_from_to(from_to, k->buf+(f<<1), t->field_defs[f].type, t->field_defs[f].size);
        rte_bv_markers_range_del(t->bv_markers+f, from_to, k->pos);

        memset(&ranges, 0, sizeof(rte_bv_ranges_t));
        ranges.bv_bs=RTE_TABLE_BV_BS;
        ranges.ranges=t->ranges[f];
        ranges.bvs=t->bvs[f];
        rte_bv_markers_to_ranges(t->bv_markers+f, 1, sizeof(uint32_t), &ranges);
        hipMemcpy(t->num_ranges+f, (void *) &ranges.num_ranges, sizeof(uint64_t), hipMemcpyHostToDevice);
    }

    return 0;
}

static int rte_table_bv_entry_add_bulk(void *t_r, void **ks_r, void **es_r, uint32_t n_keys, int *key_found, void **e_ptr) {
    struct rte_table_bv *t=(struct rte_table_bv *) t_r;
    struct rte_table_bv_key **ks=(struct rte_table_bv_key **) ks_r;

    uint32_t from_to[2];
    rte_bv_ranges_t ranges;

    for(uint32_t f=0; f<t->num_fields; ++f) {
        for(uint32_t k=0; k<n_keys; ++k) {
            cal_from_to(from_to, ks[k]->buf+(f<<1), t->field_defs[f].type, t->field_defs[f].size);
            rte_bv_markers_range_add(t->bv_markers+f, from_to, ks[k]->pos);
        }

        memset(&ranges, 0, sizeof(rte_bv_ranges_t));
        ranges.bv_bs=RTE_TABLE_BV_BS;
        ranges.ranges=t->ranges[f];
        ranges.bvs=t->bvs[f];
        rte_bv_markers_to_ranges(t->bv_markers+f, 1, sizeof(uint32_t), &ranges);
        hipMemcpy(t->num_ranges+f, (void *) &ranges.num_ranges, sizeof(uint64_t), hipMemcpyHostToDevice);
    }

    return 0;
}

static int rte_table_bv_entry_delete_bulk(void  *t_r, void **ks_r, uint32_t n_keys, int *key_found, void **es_r) {
    struct rte_table_bv *t=(struct rte_table_bv *) t_r;
    struct rte_table_bv_key **ks=(struct rte_table_bv_key **) ks_r;

    uint32_t from_to[2];
    rte_bv_ranges_t ranges;

    for(uint32_t f=0; f<t->num_fields; ++f) {
        for(uint32_t k=0; k<n_keys; ++k) {
            cal_from_to(from_to, ks[k]->buf+(f<<1), t->field_defs[f].type, t->field_defs[f].size);
            rte_bv_markers_range_del(t->bv_markers+f, from_to, ks[k]->pos);
        }

        memset(&ranges, 0, sizeof(rte_bv_ranges_t));
        ranges.bv_bs=RTE_TABLE_BV_BS;
        ranges.ranges=t->ranges[f];
        ranges.bvs=t->bvs[f];
        rte_bv_markers_to_ranges(t->bv_markers+f, 1, sizeof(uint32_t), &ranges);
        hipMemcpy(t->num_ranges+f, (void *) &ranges.num_ranges, sizeof(uint64_t), hipMemcpyHostToDevice);
    }

    return 0;
}

__global__ void bv_search(	 uint32_t **ranges,  uint64_t *num_ranges,  uint32_t *offsets,  uint8_t *sizes,
                             uint32_t *ptype_mask,  uint32_t **bvs, const uint32_t bv_bs,
                             const ulong pkts_mask, uint8_t **pkts, uint32_t *__restrict__ pkts_type,
                             volatile uint *__restrict__ positions, volatile ulong *__restrict__ lookup_hit_mask) {

    if(!((pkts_mask>>blockIdx.x)&1))
        return;

    uint8_t *pkt;
    __shared__ uint *bv[24];
    __shared__ bool field_found[24];
    uint v=0;

    field_found[threadIdx.x]=false;

    const uint32_t ptype_a=pkts_type[blockIdx.x]&ptype_mask[threadIdx.x];
    const bool ptype_matches=  (ptype_a&RTE_PTYPE_L2_MASK)!=0
                               & (ptype_a&RTE_PTYPE_L3_MASK)!=0
                               & (ptype_a&RTE_PTYPE_L4_MASK)!=0;

    if(ptype_matches) {
        pkt=pkts[blockIdx.x]+offsets[threadIdx.x];

        switch(sizes[threadIdx.x]) {
        case 1:
            v=*pkt;
            break;
        case 2:
            v=pkt[1]+(pkt[0]<<8);
            break;
        case 4:
            v=pkt[3]+(pkt[2]<<8)+(pkt[1]<<16)+(pkt[0]<<24);
            break;
        default:
            printf("[%d|%d] unknown size: %ubit\n", blockIdx.x, threadIdx.x, sizes[threadIdx.x]);
            break;
        }

        uint *range_dim=ranges[threadIdx.x];
        long long int se[]= {0, (long long int) num_ranges[threadIdx.x]};
        uint8_t l,r;
        bv[threadIdx.x]=NULL;

        for(long long int i=se[1]>>1; se[0]<=se[1]; i=(se[0]+se[1])>>1) {
            l=v>=range_dim[i<<1];
            r=v<=range_dim[(i<<1)+1];

            if(l&r) {
                bv[threadIdx.x]=bvs[threadIdx.x]+i*RTE_TABLE_BV_BS;
                field_found[threadIdx.x]=true;
                break;
            }

            se[!l]=!l?i-1:i+1;
        }
    }

    __syncthreads();
    if(!threadIdx.x) {
        uint x, pos;
        for(uint i=0; i<bv_bs; ++i) {
            x=0xffffffff;
            for(uint b=0; b<blockDim.x; ++b) {
                if(!field_found[b])
                    goto end;
                x&=bv[b][i];
            }

            if((pos=__ffs(x))!=0) {
                positions[blockIdx.x]=(i<<5)+pos-1;
                atomicOr((unsigned long long *)lookup_hit_mask, 1<<blockIdx.x);
                break;
            }
        }
    }
end:
    __syncthreads();
}

int rte_table_bv_lookup_stream(void *t_r, hipStream_t stream, struct rte_mbuf **pkts, uint64_t pkts_mask,
                               uint64_t *lookup_hit_mask, void **e) {

    struct rte_table_bv *t=(struct rte_table_bv *) t_r;

    const uint32_t n_pkts_in=__builtin_popcountll(pkts_mask);
    RTE_TABLE_BV_STATS_PKTS_IN_ADD(t, n_pkts_in);

    for(uint32_t i=0; i<n_pkts_in; ++i)
        if((pkts_mask>>i)&1) {
            t->pkts_data_h[i]=rte_pktmbuf_mtod(pkts[i], uint8_t *);
            t->packet_types_h[i]=pkts[i]->packet_type;
        }

    bv_search<<<64, t->num_fields, 0, stream>>>(	t->ranges_dev, t->num_ranges,
            t->field_offsets, t->field_sizes, t->field_ptype_masks,
            t->bvs_dev, RTE_TABLE_BV_BS,
            pkts_mask, t->pkts_data, t->packet_types,
            (uint32_t *) e, lookup_hit_mask);
    hipStreamSynchronize(stream);

    RTE_TABLE_BV_STATS_PKTS_LOOKUP_MISS(t, n_pkts_in-__builtin_popcountll(*lookup_hit_mask));
    /*
        hipError_t err = hipGetLastError();
        if(err!=hipSuccess)
            printf("[bv_search] error: %s\n", hipGetErrorString(err));
    */
    return 0;
}

static int rte_table_bv_lookup(void *t_r, struct rte_mbuf **pkts, uint64_t pkts_mask, uint64_t *lookup_hit_mask, void **e) {
    struct rte_table_bv *t=(struct rte_table_bv *) t_r;

    const uint32_t n_pkts_in=__builtin_popcountll(pkts_mask);
    RTE_TABLE_BV_STATS_PKTS_IN_ADD(t, n_pkts_in);

    for(uint32_t i=0; i<n_pkts_in; ++i)
        if((pkts_mask>>i)&1) {
            t->pkts_data_h[i]=rte_pktmbuf_mtod(pkts[i], uint8_t *);
            t->packet_types_h[i]=pkts[i]->packet_type;
        }

    bv_search<<<64, t->num_fields>>>(	t->ranges_dev, t->num_ranges,
                                        t->field_offsets, t->field_sizes, t->field_ptype_masks,
                                        t->bvs_dev, RTE_TABLE_BV_BS,
                                        pkts_mask, t->pkts_data, t->packet_types,
                                        (uint32_t *) e, lookup_hit_mask);
    hipStreamSynchronize(0);

    RTE_TABLE_BV_STATS_PKTS_LOOKUP_MISS(t, n_pkts_in-__builtin_popcountll(*lookup_hit_mask));
    /*
        hipError_t err = hipGetLastError();
        if(err!=hipSuccess)
            printf("[bv_search] error: %s\n", hipGetErrorString(err));
    */
    return 0;
}

static int rte_table_bv_stats_read(void *t_r, struct rte_table_stats *stats, int clear) {
    struct rte_table_bv *t = (struct rte_table_bv *) t_r;

    if (stats != NULL)
        memcpy(stats, &t->stats, sizeof(t->stats));

    if (clear)
        memset(&t->stats, 0, sizeof(t->stats));

    return 0;
}

struct rte_table_ops rte_table_bv_ops = {
    .f_create = rte_table_bv_create,
    .f_free = rte_table_bv_free,
    .f_add = rte_table_bv_entry_add,
    .f_delete = rte_table_bv_entry_delete,
    .f_add_bulk = rte_table_bv_entry_add_bulk,
    .f_delete_bulk = rte_table_bv_entry_delete_bulk,
    .f_lookup = rte_table_bv_lookup,
    .f_stats = rte_table_bv_stats_read
};

#ifdef __cplusplus
}
#endif
