#include "hip/hip_runtime.h"
#include "rte_table_bv.h"
#include <hip/hip_runtime.h>

#ifdef __cplusplus
extern "C" {
#endif

#include "rte_bv.h"
#include <rte_log.h>
#include <rte_malloc.h>
#include <stdlib.h>
#include <sys/time.h>
/*
#define NUM_BLOCKS 2
#define WORKERS_PER_PACKET 32
#define PACKETS_PER_BLOCK 32
*/
#define NUM_BLOCKS 64
#define WORKERS_PER_FIELD 32

#ifdef RTE_TABLE_STATS_COLLECT
#define RTE_TABLE_BV_STATS_PKTS_IN_ADD(table, val) table->stats.n_pkts_in += val
#define RTE_TABLE_BV_STATS_PKTS_LOOKUP_MISS(table, val) table->stats.n_pkts_lookup_miss += val
#else
#define RTE_TABLE_BV_STATS_PKTS_IN_ADD(table, val)
#define RTE_TABLE_BV_STATS_PKTS_LOOKUP_MISS(table, val)
#endif

struct rte_table_bv {
    uint32_t num_fields;
    struct rte_table_stats stats;
    const struct rte_table_bv_field_def *field_defs;

    uint32_t num_blocks;
    uint32_t packets_per_block;

    uint32_t ptype_mask;
    uint32_t num_rules;
    uint32_t entry_size;
    uint8_t *entries;

    uint32_t **ranges_from;
    uint32_t **ranges_to;
    uint32_t **bvs;
    uint32_t **non_zero_bvs;

    size_t *num_ranges;
    uint32_t *field_offsets;
    uint8_t *field_sizes;

    uint32_t **ranges_from_dev;
    uint32_t **ranges_to_dev;
    uint32_t **bvs_dev;
    uint32_t **non_zero_bvs_dev;

    uint8_t **pkts_data;
    uint8_t **pkts_data_h;

    uint8_t *lookup_hit_vec;
    uint8_t *lookup_hit_vec_h;

    rte_bv_markers_t *bv_markers; // size==num_fields
};

static inline int is_error(hipError_t e, const char *file, int line) {
    if(e!=hipSuccess) {
        rte_log(RTE_LOG_ERR, RTE_LOGTYPE_TABLE, "[rte_table_bv] error: %s in %s (line %d)\n", hipGetErrorString(e), file, line);
        return 1;
    }
    return 0;
}

static int rte_table_bv_free(void *t_r) {
    if(t_r==NULL)
        return 0;

    struct rte_table_bv *t=(struct rte_table_bv *) t_r;

    for(size_t i=0; i<t->num_fields; ++i) {
        hipFree(t->ranges_from[i]);
        hipFree(t->ranges_to[i]);
        hipFree(t->bvs[i]);
        hipFree(t->non_zero_bvs[i]);
    }
    hipFree(t->ranges_from_dev);
    hipFree(t->ranges_to_dev);
    hipFree(t->bvs_dev);

    hipFree(t->num_ranges);
    hipFree(t->field_offsets);
    hipFree(t->field_sizes);

    hipHostFree(t->lookup_hit_vec_h);
    hipHostFree(t->entries);
    hipHostFree(t->pkts_data_h);

    for(uint32_t i=0; i<t->num_fields; ++i)
        rte_bv_markers_free(t->bv_markers+i);

    rte_free(t->bv_markers);
    rte_free(t->ranges_from);
    rte_free(t->ranges_to);
    rte_free(t->bvs);

    rte_free(t);

    return 0;
}


static void *rte_table_bv_create(void *params, int socket_id, uint32_t entry_size) {
    struct rte_table_bv_params *p=(struct rte_table_bv_params *) params;
    struct rte_table_bv *t=(struct rte_table_bv *) rte_malloc("t", sizeof(struct rte_table_bv), 0);
    memset(t, 0, sizeof(struct rte_table_bv));

    t->num_fields=p->num_fields;
    t->packets_per_block=32/p->num_fields;
    t->num_blocks=ceil(64.0f/((double) t->packets_per_block));
    printf("packets_per_block: %u num_blocks: %u\n", t->packets_per_block, t->num_blocks);

    t->field_defs=p->field_defs;
    t->num_rules=p->num_rules;
    t->entry_size=entry_size;

    t->ranges_from=(uint32_t **) rte_malloc("ranges_from", sizeof(uint32_t *)*t->num_fields, 0);
    t->ranges_to=(uint32_t **) rte_malloc("ranges_to", sizeof(uint32_t *)*t->num_fields, 0);
    t->bvs=(uint32_t **) rte_malloc("bvs", sizeof(uint32_t *)*t->num_fields, 0);
    t->non_zero_bvs=(uint32_t **) rte_malloc("non_zero_bvs", sizeof(uint32_t *)*t->num_fields, 0);

#define IS_ERROR(X) is_error(X, __FILE__, __LINE__)
#define CHECK(X) if(IS_ERROR(X)) return NULL

    CHECK(hipHostAlloc((void **) &t->pkts_data_h, sizeof(uint8_t*)*RTE_TABLE_BV_MAX_PKTS, hipHostMallocMapped|hipHostMallocWriteCombined));
    CHECK(hipHostGetDevicePointer((void **) &t->pkts_data, t->pkts_data_h, 0));

    CHECK(hipHostAlloc((void **) &t->entries, t->entry_size*t->num_rules, hipHostMallocMapped));

    CHECK(hipHostAlloc((void **) &t->lookup_hit_vec_h, sizeof(uint8_t*)*RTE_TABLE_BV_MAX_PKTS, hipHostMallocMapped));
    CHECK(hipHostGetDevicePointer((void **) &t->lookup_hit_vec, t->lookup_hit_vec_h, 0));

    CHECK(hipMalloc((void **) &t->ranges_from_dev, sizeof(uint32_t *)*t->num_fields));
    CHECK(hipMalloc((void **) &t->ranges_to_dev, sizeof(uint32_t *)*t->num_fields));
    CHECK(hipMalloc((void **) &t->bvs_dev, sizeof(uint32_t *)*t->num_fields));
    CHECK(hipMalloc((void **) &t->non_zero_bvs_dev, sizeof(uint32_t *)*t->num_fields));
    CHECK(hipMalloc((void **) &t->field_offsets, sizeof(uint32_t)*t->num_fields));
    CHECK(hipMalloc((void **) &t->field_sizes, sizeof(uint32_t)*t->num_fields));
    CHECK(hipMalloc((void **) &t->num_ranges, sizeof(uint64_t)*t->num_fields));

    t->ptype_mask=UINT32_MAX;

    for(size_t i=0; i<t->num_fields; ++i) {
        CHECK(hipMemcpy(t->field_offsets+i, &t->field_defs[i].offset, sizeof(uint32_t), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(t->field_sizes+i, &t->field_defs[i].size, sizeof(uint32_t), hipMemcpyHostToDevice));
        t->ptype_mask&=t->field_defs[i].ptype_mask;

        CHECK(hipMalloc((void **) &t->ranges_from[i], sizeof(uint32_t)*((size_t) RTE_TABLE_BV_MAX_RANGES)));
        CHECK(hipMalloc((void **) &t->ranges_to[i], sizeof(uint32_t)*((size_t) RTE_TABLE_BV_MAX_RANGES)));
        printf("size: bvs[%ld] %ld bytes\n", i, sizeof(uint32_t)*((size_t) RTE_TABLE_BV_BS) * ((size_t ) RTE_TABLE_BV_MAX_RANGES));
        CHECK(hipMalloc((void **) &t->bvs[i], sizeof(uint32_t)*((size_t) RTE_TABLE_BV_BS) * ((size_t ) RTE_TABLE_BV_MAX_RANGES)));
        printf("size: non_zero_bvs[%ld] %ld bytes\n", i, sizeof(uint32_t)*((size_t) RTE_TABLE_BV_BS>>5) * ((size_t ) RTE_TABLE_BV_MAX_RANGES));
        CHECK(hipMalloc((void **) &t->non_zero_bvs[i], sizeof(uint32_t)*((size_t) RTE_TABLE_BV_BS>>5) * ((size_t ) RTE_TABLE_BV_MAX_RANGES)));
    }

    CHECK(hipMemcpy(t->ranges_from_dev, t->ranges_from, sizeof(uint32_t *)*t->num_fields, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(t->ranges_to_dev, t->ranges_to, sizeof(uint32_t *)*t->num_fields, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(t->bvs_dev, t->bvs, sizeof(uint32_t *)*t->num_fields, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(t->non_zero_bvs_dev, t->non_zero_bvs, sizeof(uint32_t *)*t->num_fields, hipMemcpyHostToDevice));
    CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

#undef CHECK
#undef IS_ERROR

    t->bv_markers=(rte_bv_markers_t *) rte_malloc("bv_markers", sizeof(rte_bv_markers_t)*t->num_fields, 0);

    for(size_t i=0; i<t->num_fields; ++i) {
        if(rte_bv_markers_create(&t->bv_markers[i])) {
            rte_table_bv_free(t);
            rte_log(RTE_LOG_ERR, RTE_LOGTYPE_TABLE, "Error creating marker!\n");
            return NULL;
        }
    }

    return (void *) t;
}
#undef IS_ERROR

static inline void cal_from_to(uint32_t *from_to, uint32_t *v, uint8_t type, uint8_t size) {
    if(type==RTE_TABLE_BV_FIELD_TYPE_RANGE) {
        from_to[0]=*v;
        from_to[1]=v[1];
    } else {
        from_to[0]=(*v)&v[1];
        switch(size) {
        case 1:
            from_to[1]=(*v)|((uint8_t) (~v[1]));
            break;
        case 2:
            from_to[1]=(*v)|((uint16_t) (~v[1]));
            break;
        case 4:
            from_to[1]=(*v)|((uint32_t) (~v[1]));
            break;
        default:
#ifdef DEBUG
            fprintf(stderr, "[cal_from_to] error: unknown size: %d bits\n", size);
#endif
            break;
        }
    }
}

static int rte_table_bv_entry_add(void *t_r, void *k_r, void *e_r, int *key_found, __rte_unused void **e_ptr) {
    struct rte_table_bv *t=(struct rte_table_bv *) t_r;
    struct rte_table_bv_key *k=(struct rte_table_bv_key *) k_r;

    if(key_found)
        *key_found=0;

    uint32_t from_to[2];
    rte_bv_ranges_t ranges;

    for(uint32_t f=0; f<t->num_fields; ++f) {
        cal_from_to(from_to, k->buf +(f<<1), t->field_defs[f].type, t->field_defs[f].size);
        rte_bv_markers_range_add(t->bv_markers+f, from_to, k->pos);

        memset(&ranges, 0, sizeof(rte_bv_ranges_t));
        ranges.bv_bs=RTE_TABLE_BV_BS;
        ranges.max_num_ranges=RTE_TABLE_BV_MAX_RANGES;
        ranges.ranges_from=t->ranges_from[f];
        ranges.ranges_to=t->ranges_to[f];
        ranges.bvs=t->bvs[f];
        ranges.non_zero_bvs=t->non_zero_bvs[f];
        if(rte_bv_markers_to_ranges(t->bv_markers+f, 1, sizeof(uint32_t), &ranges))
            return 1;
        hipMemcpy(t->num_ranges+f, (void *) &ranges.num_ranges, sizeof(uint64_t), hipMemcpyHostToDevice);
    }

    hipMemcpy(&t->entries[t->entry_size*k->pos], e_r, t->entry_size, hipMemcpyHostToDevice);

    if(e_ptr)
        *e_ptr=&t->entries[t->entry_size*k->pos];

    return 0;
}

static int rte_table_bv_entry_delete(void  *t_r, void *k_r, int *key_found, __rte_unused void *e) {
    struct rte_table_bv *t=(struct rte_table_bv *) t_r;
    struct rte_table_bv_key *k=(struct rte_table_bv_key *) k_r;

    if(key_found)
        *key_found=0;

    uint32_t from_to[2];
    rte_bv_ranges_t ranges;

    for(uint32_t f=0; f<t->num_fields; ++f) {
        cal_from_to(from_to, k->buf+(f<<1), t->field_defs[f].type, t->field_defs[f].size);
        rte_bv_markers_range_del(t->bv_markers+f, from_to, k->pos);

        memset(&ranges, 0, sizeof(rte_bv_ranges_t));
        ranges.bv_bs=RTE_TABLE_BV_BS;
        ranges.max_num_ranges=RTE_TABLE_BV_MAX_RANGES;
        ranges.ranges_from=t->ranges_from[f];
        ranges.ranges_to=t->ranges_to[f];
        ranges.bvs=t->bvs[f];
        ranges.non_zero_bvs=t->non_zero_bvs[f];
        if(rte_bv_markers_to_ranges(t->bv_markers+f, 1, sizeof(uint32_t), &ranges))
            return 1;
        hipMemcpy(t->num_ranges+f, (void *) &ranges.num_ranges, sizeof(uint64_t), hipMemcpyHostToDevice);
    }

    return 0;
}

static int rte_table_bv_entry_add_bulk(void *t_r, void **ks_r, void **es_r, uint32_t n_keys, int *key_found, __rte_unused void **e_ptr) {
    struct rte_table_bv *t=(struct rte_table_bv *) t_r;
    struct rte_table_bv_key **ks=(struct rte_table_bv_key **) ks_r;

    uint32_t from_to[2];
    rte_bv_ranges_t ranges;

    for(uint32_t f=0; f<t->num_fields; ++f) {
        for(uint32_t k=0; k<n_keys; ++k) {
            cal_from_to(from_to, ks[k]->buf+(f<<1), t->field_defs[f].type, t->field_defs[f].size);
            rte_bv_markers_range_add(t->bv_markers+f, from_to, ks[k]->pos);
        }

        memset(&ranges, 0, sizeof(rte_bv_ranges_t));
        ranges.bv_bs=RTE_TABLE_BV_BS;
        ranges.max_num_ranges=RTE_TABLE_BV_MAX_RANGES;
        ranges.ranges_from=t->ranges_from[f];
        ranges.ranges_to=t->ranges_to[f];
        ranges.bvs=t->bvs[f];
        ranges.non_zero_bvs=t->non_zero_bvs[f];
        if(rte_bv_markers_to_ranges(t->bv_markers+f, 1, sizeof(uint32_t), &ranges))
            return 1;
        hipMemcpy(t->num_ranges+f, (void *) &ranges.num_ranges, sizeof(uint64_t), hipMemcpyHostToDevice);
    }


    for(uint32_t k=0; k<n_keys; ++k) {
        if(key_found)
            key_found[k]=0;

        hipMemcpy(&t->entries[t->entry_size*ks[k]->pos], es_r[ks[k]->pos], t->entry_size, hipMemcpyHostToDevice);

        if(e_ptr)
            e_ptr[k]=&t->entries[t->entry_size*ks[k]->pos];
    }

    return 0;
}

static int rte_table_bv_entry_delete_bulk(void  *t_r, void **ks_r, uint32_t n_keys, int *key_found, __rte_unused void **es_r) {
    struct rte_table_bv *t=(struct rte_table_bv *) t_r;
    struct rte_table_bv_key **ks=(struct rte_table_bv_key **) ks_r;

    uint32_t from_to[2];
    rte_bv_ranges_t ranges;

    for(uint32_t f=0; f<t->num_fields; ++f) {
        for(uint32_t k=0; k<n_keys; ++k) {
            cal_from_to(from_to, ks[k]->buf+(f<<1), t->field_defs[f].type, t->field_defs[f].size);
            rte_bv_markers_range_del(t->bv_markers+f, from_to, ks[k]->pos);
        }

        memset(&ranges, 0, sizeof(rte_bv_ranges_t));
        ranges.bv_bs=RTE_TABLE_BV_BS;
        ranges.max_num_ranges=RTE_TABLE_BV_MAX_RANGES;
        ranges.ranges_from=t->ranges_from[f];
        ranges.ranges_to=t->ranges_to[f];
        ranges.bvs=t->bvs[f];
        ranges.non_zero_bvs=t->non_zero_bvs[f];
        if(rte_bv_markers_to_ranges(t->bv_markers+f, 1, sizeof(uint32_t), &ranges))
            return 1;
        hipMemcpy(t->num_ranges+f, (void *) &ranges.num_ranges, sizeof(uint64_t), hipMemcpyHostToDevice);
    }

    if(key_found)
        for(uint32_t k=0; k<n_keys; ++k)
            key_found[k]=0;

    return 0;
}

__global__ void bv_search(	uint32_t *__restrict__ *__restrict__ ranges_from,
                            uint32_t *__restrict__ *__restrict__ ranges_to,
                            const uint64_t *__restrict__ num_ranges,
                            const uint32_t *__restrict__ offsets, const uint8_t *__restrict__ sizes,
                            uint32_t *__restrict__ *__restrict__ bvs, uint32_t *__restrict__ *__restrict__ non_zero_bvs,
                            const uint32_t num_fields,
                            const uint32_t entry_size, uint8_t *__restrict__ entries,
                            const uint64_t pkts_mask, uint8_t *__restrict__ *__restrict__ pkts,
                            void *__restrict__ *matched_entries, uint8_t *__restrict__ lookup_hit_vec) {

#define field_id threadIdx.z

    int pkt_id=blockDim.y*blockIdx.x+threadIdx.y;
    __shared__ uint *__restrict__ bv[32][RTE_TABLE_BV_MAX_FIELDS];
    __shared__ uint *__restrict__ non_zero_bv[32][RTE_TABLE_BV_MAX_FIELDS];
    __shared__ uint32_t bv_not_found[32];

    if(!((pkts_mask>>pkt_id)&1LU))
        return;

    uint v;
    if(!threadIdx.x) {
        bv_not_found[threadIdx.y]=0;
        bv[threadIdx.y][field_id]=NULL;
		const uint8_t *pkt=(uint8_t * ) pkts[pkt_id]+offsets[field_id];
		switch(sizes[field_id]) {
        case 1:
            v=*pkt;
            break;
        case 2:
            v=pkt[1]|(pkt[0]<<8);
            break;
        case 4:
            v=pkt[3]|(pkt[2]<<8)|(pkt[1]<<16)|(pkt[0]<<24);
            break;
        default:
            __builtin_unreachable();
            printf("[%d|%d] unknown size: %u byte\n", blockIdx.x, threadIdx.y, sizes[field_id]);
            break;
        }
    }
    __syncwarp();
    v=__shfl_sync(UINT32_MAX, v, 0);
    long size=num_ranges[field_id]>>5;
    long start=0, offset;
    uint32_t l,r; //left, right

    while(size) {
        offset=start+((long) threadIdx.x)*size;
        l=__ballot_sync(UINT32_MAX, v>=ranges_from[field_id][offset]);
        r=__ballot_sync(UINT32_MAX, v<=ranges_to[field_id][offset]);
        if(l&r) {
            if((__ffs(l&r)-1)==threadIdx.x) {
//				printf("[%d|%d] %08X <= %08X <= %08X, %ld\n", pkt_id, field_id, ranges_from[field_id][offset], v, ranges_to[field_id][offset], offset);
                bv[threadIdx.y][field_id]=bvs[field_id]+offset*RTE_TABLE_BV_BS;
                non_zero_bv[threadIdx.y][field_id]=non_zero_bvs[field_id]+offset*RTE_TABLE_NON_ZERO_BV_BS;
            }
            goto found_bv;
        }
        if(!l)
            goto found_bv;

        //reuse r to save one register per thread
        r=__popc(l)-1;
        start=__shfl_sync(UINT32_MAX, offset+1, r);
        size=r==31?(num_ranges[field_id]-start)>>5:(size-1)>>5;

        __syncwarp();
    }
    offset=start+threadIdx.x;
    l=__ballot_sync(UINT32_MAX, offset<num_ranges[field_id]?v>=ranges_from[field_id][offset]:0);
    r=__ballot_sync(UINT32_MAX, offset<num_ranges[field_id]?v<=ranges_to[field_id][offset]:0);
    if(l&r) {
        if((__ffs(l&r)-1)==threadIdx.x) {
            //printf("[%d|%d] %08X <= %08X <= %08X, %ld\n", pkt_id, field_id, ranges_from[field_id][offset], v, ranges_to[field_id][offset], offset);
            bv[threadIdx.y][field_id]=bvs[field_id]+offset*RTE_TABLE_BV_BS;
            non_zero_bv[threadIdx.y][field_id]=non_zero_bvs[field_id]+offset*RTE_TABLE_NON_ZERO_BV_BS;
        }
    }

found_bv:

    if(!threadIdx.x && !bv[threadIdx.y][field_id]) {
        bv_not_found[threadIdx.y]=1;
    }

    __syncthreads();

    if(bv_not_found[threadIdx.y])
        return;

    if(threadIdx.z!=0)
        return;

#undef field_id

    __syncwarp();

    uint32_t in_loop;
    // all bitvectors found, now getting highest-priority rule
    uint x, y, tm;
    for(int nz_bv_b=threadIdx.x; nz_bv_b<RTE_TABLE_NON_ZERO_BV_BS; nz_bv_b+=blockDim.x) { // TODO maybe use WORKERS_PER_PACKET
        in_loop=__activemask();
        x=UINT32_MAX;

        for(int field_id=0; field_id<num_fields; ++field_id)
            x&=non_zero_bv[threadIdx.y][field_id][nz_bv_b];

        int pos;
        while((pos=__ffs(x))) {
            y=UINT32_MAX;
            for(int field_id=0; field_id<num_fields; ++field_id)
                y&=bv[threadIdx.y][field_id][(nz_bv_b<<5)|(pos-1)];
            if(y)
                break;
            x=(x>>pos)<<pos;
        }

        __syncwarp(in_loop); //TODO maybe remove
        if((tm=__ballot_sync(in_loop, __ffs(y)))) {
            if((__ffs(tm)-1)==threadIdx.x) {
                matched_entries[pkt_id]=(void *) &entries[entry_size*((nz_bv_b<<5)+__ffs(y)-1LU)];
                lookup_hit_vec[pkt_id]=1;
                //atomicOr((unsigned long long int *)lookup_hit_mask, 1LU<<pkt_id);
                //__threadfence_system();
            }
            break;
        }
    }

}


#define IS_ERROR(X) is_error(X, __FILE__, __LINE__)
int rte_table_bv_lookup_stream(void *t_r, hipStream_t stream, struct rte_mbuf **pkts, uint64_t pkts_mask,
                               uint64_t *lookup_hit_mask, void **e) {
#ifdef MEASURE_TIME
    struct timeval k_t1,k_t2,l_t1,l_t2;
    gettimeofday(&l_t1, NULL);
#endif

    struct rte_table_bv *t=(struct rte_table_bv *) t_r;

    const uint32_t n_pkts_in=__builtin_popcountll(pkts_mask);
    RTE_TABLE_BV_STATS_PKTS_IN_ADD(t, n_pkts_in);

    uint64_t real_pkts_mask=0;
    for(uint32_t i=0; i<n_pkts_in; ++i) {
        const uint32_t mp=pkts[i]->packet_type&t->ptype_mask;
        if((pkts_mask>>i)&1&((mp&RTE_PTYPE_L2_MASK)!=0)&((mp&RTE_PTYPE_L3_MASK)!=0)&((mp&RTE_PTYPE_L4_MASK)!=0)) {
            t->pkts_data_h[i]=rte_pktmbuf_mtod(pkts[i], uint8_t *);
            real_pkts_mask|=1LU<<i;
        }
    }

#ifdef MEASURE_TIME
    gettimeofday(&k_t1, NULL);
#endif

    bv_search<<<t->num_blocks, dim3{WORKERS_PER_FIELD, t->packets_per_block, t->num_fields}, 0, stream>>>(	t->ranges_from_dev, t->ranges_to_dev, t->num_ranges,
            t->field_offsets, t->field_sizes,
            t->bvs_dev, t->non_zero_bvs_dev, t->num_fields, t->entry_size, t->entries,
            real_pkts_mask, t->pkts_data,
            e, t->lookup_hit_vec);

    hipStreamSynchronize(stream);

#ifdef MEASURE_TIME
    gettimeofday(&k_t2, NULL);
    printf("KERNEL took %luus\n", (k_t2.tv_sec*1000000+k_t2.tv_usec)-(k_t1.tv_sec*1000000+k_t1.tv_usec));
#endif
    uint64_t lhm=0;
    for(uint32_t i=0; i<n_pkts_in; ++i) {
        if(t->lookup_hit_vec_h[i]) {
            lhm|=1LU<<i;
            t->lookup_hit_vec_h[i]=0;
        }
    }
    *lookup_hit_mask=lhm;

    RTE_TABLE_BV_STATS_PKTS_LOOKUP_MISS(t, n_pkts_in-__builtin_popcountll(*lookup_hit_mask));

#ifdef MEASURE_TIME
    gettimeofday(&k_t2, NULL);
    gettimeofday(&l_t2, NULL);
    printf("LOOKUP took %luus\n", (l_t2.tv_sec*1000000+l_t2.tv_usec)-(l_t1.tv_sec*1000000+l_t1.tv_usec));
#endif
    return 0;
}

static int rte_table_bv_lookup(void *t_r, struct rte_mbuf **pkts, uint64_t pkts_mask, uint64_t *lookup_hit_mask, void **e) {
    struct rte_table_bv *t=(struct rte_table_bv *) t_r;

    const uint32_t n_pkts_in=__builtin_popcountll(pkts_mask);
    RTE_TABLE_BV_STATS_PKTS_IN_ADD(t, n_pkts_in);

    uint64_t real_pkts_mask=0;
    for(uint32_t i=0; i<n_pkts_in; ++i) {
        const uint32_t mp=pkts[i]->packet_type&t->ptype_mask;
        if((pkts_mask>>i)&1&((mp&RTE_PTYPE_L2_MASK)!=0)&((mp&RTE_PTYPE_L3_MASK)!=0)&((mp&RTE_PTYPE_L4_MASK)!=0)) {
            t->pkts_data_h[i]=rte_pktmbuf_mtod(pkts[i], uint8_t *);
            real_pkts_mask|=1LU<<i;
        }
    }

    bv_search<<<NUM_BLOCKS, dim3{WORKERS_PER_FIELD, t->num_fields}>>>(	t->ranges_from_dev, t->ranges_to_dev, t->num_ranges,
            t->field_offsets, t->field_sizes,
            t->bvs_dev, t->non_zero_bvs_dev, t->num_fields, t->entry_size, t->entries,
            real_pkts_mask, t->pkts_data,
            e, t->lookup_hit_vec);

    hipStreamSynchronize(0);

    uint64_t lhm=0;
    for(uint32_t i=0; i<n_pkts_in; ++i) {
        if(t->lookup_hit_vec_h[i]) {
            lhm|=1LU<<i;
            t->lookup_hit_vec_h[i]=0;
        }
    }
    *lookup_hit_mask=lhm;


    RTE_TABLE_BV_STATS_PKTS_LOOKUP_MISS(t, n_pkts_in-__builtin_popcountll(*lookup_hit_mask));

    return 0;
}

static int rte_table_bv_stats_read(void *t_r, struct rte_table_stats *stats, int clear) {
    struct rte_table_bv *t = (struct rte_table_bv *) t_r;

    if (stats != NULL)
        memcpy(stats, &t->stats, sizeof(t->stats));

    if (clear)
        memset(&t->stats, 0, sizeof(t->stats));

    return 0;
}

struct rte_table_ops rte_table_bv_ops = {
    .f_create = rte_table_bv_create,
    .f_free = rte_table_bv_free,
    .f_add = rte_table_bv_entry_add,
    .f_delete = rte_table_bv_entry_delete,
    .f_add_bulk = rte_table_bv_entry_add_bulk,
    .f_delete_bulk = rte_table_bv_entry_delete_bulk,
    .f_lookup = rte_table_bv_lookup,
    .f_stats = rte_table_bv_stats_read
};

#ifdef __cplusplus
}
#endif
